#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "cuda_nbody.h"
#include <stdio.h>

// functions to move info from cpu to gpu and vice versa
__host__ ParticleGPU particleToGPU(const Particle &p)
{
    ParticleGPU gpuP;
    gpuP.position = make_float4(p.position.x, p.position.y, p.position.z, p.position.w);
    gpuP.velocity = make_float4(p.velocity.x, p.velocity.y, p.velocity.z, p.velocity.w);
    gpuP.acceleration = make_float4(p.acceleration.x, p.acceleration.y, p.acceleration.z, p.acceleration.w);
    gpuP.mass = p.mass;
    return gpuP;
}

__host__ void gpuToParticle(const ParticleGPU &gpuP, Particle &p)
{
    p.position = glm::vec4(gpuP.position.x, gpuP.position.y, gpuP.position.z, gpuP.position.w);
    p.velocity = glm::vec4(gpuP.velocity.x, gpuP.velocity.y, gpuP.velocity.z, gpuP.velocity.w);
    p.acceleration = glm::vec4(gpuP.acceleration.x, gpuP.acceleration.y, gpuP.acceleration.z, gpuP.acceleration.w);
    p.mass = gpuP.mass;
}

// same stuff as physics file but using kernels

__global__ void resetAccelerationKernel(ParticleGPU *particles, int numParticles)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numParticles)
    {
        particles[idx].acceleration = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }
}

__global__ void computeForcesKernel(ParticleGPU *particles, int numParticles, float G, float softening)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles)
    {
        float4 myPos = particles[idx].position;
        float myMass = particles[idx].mass;
        float4 acc = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        if (idx == 0)
        {
            particles[idx].acceleration = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            return;
        }

        // black holes the first particle
        float4 bhPos = particles[0].position;
        float bhMass = particles[0].mass;

        float3 direction;
        direction.x = bhPos.x - myPos.x;
        direction.y = bhPos.y - myPos.y;
        direction.z = bhPos.z - myPos.z;

        float distSquared = direction.x * direction.x +
                            direction.y * direction.y +
                            direction.z * direction.z +
                            softening;

        float dist = sqrtf(distSquared);
        float invDist = 1.0f / dist;

        direction.x *= invDist;
        direction.y *= invDist;
        direction.z *= invDist;

        float forceMag = G * myMass * bhMass / distSquared;

        acc.x += direction.x * forceMag / myMass;
        acc.y += direction.y * forceMag / myMass;
        acc.z += direction.z * forceMag / myMass;

        // Calculate forces from all other particles
        for (int j = 1; j < numParticles; j++)
        {
            if (j == idx)
                continue;

            float4 otherPos = particles[j].position;
            float otherMass = particles[j].mass;

            float3 dirOther;
            dirOther.x = otherPos.x - myPos.x;
            dirOther.y = otherPos.y - myPos.y;
            dirOther.z = otherPos.z - myPos.z;

            float distSqOther = dirOther.x * dirOther.x +
                                dirOther.y * dirOther.y +
                                dirOther.z * dirOther.z +
                                softening;

            float distOther = sqrtf(distSqOther);
            float invDistOther = 1.0f / distOther;

            dirOther.x *= invDistOther;
            dirOther.y *= invDistOther;
            dirOther.z *= invDistOther;

            float forceMagOther = G * myMass * otherMass / distSqOther;

            acc.x += dirOther.x * forceMagOther / myMass;
            acc.y += dirOther.y * forceMagOther / myMass;
            acc.z += dirOther.z * forceMagOther / myMass;
        }

        // left remaining implementation same
        float maxAcc = 1000.0f;
        float accMag = sqrtf(acc.x * acc.x + acc.y * acc.y + acc.z * acc.z);

        if (accMag > maxAcc)
        {
            float scale = maxAcc / accMag;
            acc.x *= scale;
            acc.y *= scale;
            acc.z *= scale;
        }

        particles[idx].acceleration = acc;
    }
}

// CUDA part 1 for integrate leap frog
__global__ void integrateLeapFrogKernel1(ParticleGPU *particles, int numParticles, float dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // implementation practically is the same

    if (idx < numParticles)
    {
        if (idx == 0)
            return;

        float4 vel = particles[idx].velocity;
        float4 acc = particles[idx].acceleration;

        vel.x += acc.x * dt * 0.5f;
        vel.y += acc.y * dt * 0.5f;
        vel.z += acc.z * dt * 0.5f;

        particles[idx].position.x += vel.x * dt;
        particles[idx].position.y += vel.y * dt;
        particles[idx].position.z += vel.z * dt;

        particles[idx].velocity = vel;
    }
}

// CUDA part 2 for leap-frog integration
__global__ void integrateLeapFrogKernel2(ParticleGPU *particles, int numParticles, float dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles)
    {
        // pretty much the same as before
        if (idx == 0)
            return;

        float4 vel = particles[idx].velocity;
        float4 acc = particles[idx].acceleration;

        vel.x += acc.x * dt * 0.5f;
        vel.y += acc.y * dt * 0.5f;
        vel.z += acc.z * dt * 0.5f;

        float3 pos;
        pos.x = particles[idx].position.x;
        pos.y = particles[idx].position.y;
        pos.z = particles[idx].position.z;

        float distFromCenter = sqrtf(pos.x * pos.x + pos.y * pos.y + pos.z * pos.z);
        if (distFromCenter > 30.0f)
        {
            vel.x *= 0.998f;
            vel.y *= 0.998f;
            vel.z *= 0.998f;
        }

        particles[idx].velocity = vel;
    }
}

void allocateParticlesGPU(ParticleGPU **devParticles, int numParticles)
{
    CUDA_CHECK_ERROR(hipMalloc((void **)devParticles, numParticles * sizeof(ParticleGPU)));
}

void freeParticlesGPU(ParticleGPU *devParticles)
{
    if (devParticles)
    {
        CUDA_CHECK_ERROR(hipFree(devParticles));
    }
}

void copyParticlesToGPU(ParticleGPU *devParticles, const Particle *hostParticles, int numParticles)
{
    // temp particles to fill in real info
    ParticleGPU *tempGPUParticles = new ParticleGPU[numParticles];

    for (int i = 0; i < numParticles; i++)
    {
        tempGPUParticles[i] = particleToGPU(hostParticles[i]);
    }

    // Copy to device
    CUDA_CHECK_ERROR(hipMemcpy(devParticles, tempGPUParticles,
                                numParticles * sizeof(ParticleGPU),
                                hipMemcpyHostToDevice));

    delete[] tempGPUParticles;
}

void copyParticlesFromGPU(Particle *hostParticles, const ParticleGPU *devParticles, int numParticles)
{
    // temp particles
    ParticleGPU *tempGPUParticles = new ParticleGPU[numParticles];

    // Copy from device
    CUDA_CHECK_ERROR(hipMemcpy(tempGPUParticles, devParticles,
                                numParticles * sizeof(ParticleGPU),
                                hipMemcpyDeviceToHost));

    for (int i = 0; i < numParticles; i++)
    {
        gpuToParticle(tempGPUParticles[i], hostParticles[i]);
    }

    delete[] tempGPUParticles;
}

void computeGravitationalForcesGPU(ParticleGPU *devParticles, int numParticles, float G, float softening)
{
    int blockSize = 512;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;

    resetAccelerationKernel<<<numBlocks, blockSize>>>(devParticles, numParticles);

    computeForcesKernel<<<numBlocks, blockSize>>>(devParticles, numParticles, G, softening);

    // Check for kernel errors
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
}

void integrateParticlesGPU(ParticleGPU *devParticles, int numParticles, float dt)
{
    // Set grid and block dimensions
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;

    // First half of leap-frog integration
    integrateLeapFrogKernel1<<<numBlocks, blockSize>>>(devParticles, numParticles, dt);

    // Compute forces (this will be called separately)

    // Second half of leap-frog integration
    integrateLeapFrogKernel2<<<numBlocks, blockSize>>>(devParticles, numParticles, dt);

    // Check for kernel errors
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
}
